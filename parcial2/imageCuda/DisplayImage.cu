#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <opencv2/opencv.hpp>

using namespace cv;
using namespace std;

#define chSize 3

#define N 10

__global__ void Matriz_GPU_Mult(double *a, double *b, double *c) {
	int k, sum = 0;
	int i = blockIdx.x * blockDim.x + threadIdx.x; 
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < N && j < N) {
    for (k = 0; k < N; k++) {
      sum += a[j * N + k] * b[k * N + i];
    }
    c[j * N + i] = sum;
  }
}

/*
__global__ void gpuGrayScale(int *A, float *B, int cols, int rows){
  int tidx = (blockDim.x * blockIdx.x + threadIdx.x) + chSize;
  int tidy = blockDim.y * blockIdx.y + threadIdx.y;

  float r,g,b;

  printf("%d,%d", tidx, tidy);

  for(int row = tidy; row < rows; row++){
    for(int col = tidx; col < cols; col += chSize){
      r = A[row * cols + col];
      g = A[row * cols + col + 1];
      b = A[row * cols + col + 2];
      
      for(int k = chSize - 1; k >= 0; k--){
        B[row * cols + col - k] = (r * 0.299 + g * 0.587 + b * 0.114);
      }
    }
  }

}
*/
int main(int argc, char** argv )
{
  double timeGPU; //, timeCPU;
  double A[N][N], B[N][N], C[N][N];
  double *d_a, *d_b, *d_c;
  int cont,i,j;

  //inicializacion
  for (i = 0; i < N; i++) {
    cont = 0;
    for (j = 0; j < N; j++) {
      A[i][j] = cont * 987;
      B[i][j] = cont * 987;
      cont++;
    }
  }

  size_t bytes = N * sizeof(double);

  hipMalloc((void **) &d_a, bytes);
  hipMalloc((void **) &d_b, bytes);
  hipMalloc((void **) &d_c, bytes);

  hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

  //int threadsPerBlock(16);
  //int numBlocks(N/threadsPerBlock);
  dim3 threadsPerBlock(3, 3);
  dim3 numBlocks((int)ceil((float)N/threadsPerBlock.x), (int)ceil((float)N/threadsPerBlock.y));
  
  clock_t startGPU  = clock();
  Matriz_GPU_Mult<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);
  timeGPU = ((double)(clock() - startGPU))/CLOCKS_PER_SEC;
  
  hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);
  
  /*
  clock_t startCPU = clock();
  Matriz_CPU_Mult(A, B, C);
  timeCPU = ((double)(clock() - startCPU))/CLOCKS_PER_SEC;
  */

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // tiempos de ejecucion
  printf("tiempo GPU = %f s\n",timeGPU);
  cout << C[0] << endl;

/*
  if ( argc != 2 )
  {
    printf("usage: DisplayImage.out <Image_Path>\n");
    return -1;
  }

  Mat image;
  image = imread( argv[1], 1 );

  if ( !image.data )
  {
    printf("No image data \n");
    return -1;
  }

  int *h_a, *d_a;
  float *h_b, *d_b;
  int img_size = image.rows * image.cols;

  h_a = (int *)malloc(img_size * sizeof(int));
  h_b = (float *)malloc(img_size * sizeof(float));

  hipMalloc((void **) &d_a, img_size * sizeof(int));
  hipMalloc((void **) &d_b, img_size * sizeof(float));

  hipMemcpy(d_a, h_a, img_size * sizeof(int), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(32, 32);
  dim3 numBlocks((int)ceil((float)image.cols/threadsPerBlock.x), (int)ceil((float)image.rows/threadsPerBlock.y));

  gpuGrayScale<<<numBlocks, threadsPerBlock>>>(d_a, d_b, image.cols, image.rows);
  cout << "im here" << endl;
  hipMemcpy(h_b, d_b, img_size, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);

  //namedWindow("Display Image", WINDOW_AUTOSIZE );
  //imshow("Display Image", image);
  
  //Mat img = (Mat_<float>(image.rows, image.cols) << h_b);
  //img = h_b;
  /*
  float r,g,b;
  for(int y=0;y<image.rows;y++){
    for(int x=0;x<image.cols;x++){
      // get pixel
      Vec3b color = img.at<Vec3b>(Point(x,y));

      r = color[0];
      g = color[1];
      b = color[2];

      //I = .299f * R + .587f * G + .114f * B
      color[2] = (r * 0.299 + g * 0.587 + b * 0.114);
      color[1] = (r * 0.299 + g * 0.587 + b * 0.114);
      color[0] = (r * 0.299 + g * 0.587 + b * 0.114);

      // set pixel
      img.at<Vec3b>(Point(x,y)) = color;
    }
  }
  */
  //imwrite("lena_out.jpg", img);

  //cout << h_b[0] << endl;

  //waitKey(0);

  return 0;
}