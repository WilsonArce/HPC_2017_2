#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <opencv2/opencv.hpp>

using namespace cv;
using namespace std;

#define chSize 3

#define MWIDTH 512
#define MTILE 16
#define BWIDTH 16

__global__ void gpu_matrixMul(int *a, int *b, int *c, int Width, int tile_width){

  int start_row = (blockDim.y*blockIdx.y + threadIdx.y)*tile_width;
  int end_row = start_row + tile_width;
  int start_col = (blockDim.x*blockIdx.x + threadIdx.x)*tile_width;
  int end_col = start_col + tile_width;

  for (int row = start_row; row < end_row; row++) {
    for(int col = start_col; col < end_col; col++) {
      float sum = 0;
      for (int k = 0; k < Width; k++) {
        sum += a[row * Width + k]*b[k * Width + col];
      }
      c[row*Width+col] = sum;
    }
  }
}

/*
__global__ void gpuGrayScale(int *A, float *B, int cols, int rows){
  int tidx = (blockDim.x * blockIdx.x + threadIdx.x) + chSize;
  int tidy = blockDim.y * blockIdx.y + threadIdx.y;

  float r,g,b;

  printf("%d,%d", tidx, tidy);

  for(int row = tidy; row < rows; row++){
    for(int col = tidx; col < cols; col += chSize){
      r = A[row * cols + col];
      g = A[row * cols + col + 1];
      b = A[row * cols + col + 2];
      
      for(int k = chSize - 1; k >= 0; k--){
        B[row * cols + col - k] = (r * 0.299 + g * 0.587 + b * 0.114);
      }
    }
  }

}
*/
int main(int argc, char** argv )
{
  double timeGPU;
  int *h_a, *h_b, *h_c, *d_a, *d_b, *d_c;
  h_a = (int *)malloc(MWIDTH*MWIDTH*sizeof(int));
  h_b = (int *)malloc(MWIDTH*MWIDTH*sizeof(int));
  h_c = (int *)malloc(MWIDTH*MWIDTH*sizeof(int));
  hipMalloc(&d_a, MWIDTH*MWIDTH*sizeof(int));
  hipMalloc(&d_b, MWIDTH*MWIDTH*sizeof(int));
  hipMalloc(&d_c, MWIDTH*MWIDTH*sizeof(int));

  for (int i = 0; i < MWIDTH*MWIDTH; i++) {
    h_a[i] = 1;
    h_b[i] = 1;
    h_c[i] = 0;}

  hipMemcpy(d_a, h_a, MWIDTH*MWIDTH*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, MWIDTH*MWIDTH*sizeof(int), hipMemcpyHostToDevice);
  hipMemset(d_c, 0, MWIDTH*MWIDTH*sizeof(int));

  clock_t startGPU  = clock();
  gpu_matrixMul<<<dim3((MWIDTH/(MTILE*BWIDTH)), (MWIDTH/(MTILE*BWIDTH))), dim3(BWIDTH,BWIDTH)>>>(d_a, d_b, d_c, MWIDTH, MTILE);

  hipMemcpy(h_c, d_c, MWIDTH*MWIDTH*sizeof(int), hipMemcpyDeviceToHost);
  timeGPU = ((double)(clock() - startGPU))/CLOCKS_PER_SEC;

  printf("tiempo GPU = %f s\n",timeGPU);
  cout << h_c[9][9] << endl;

/*
  if ( argc != 2 )
  {
    printf("usage: DisplayImage.out <Image_Path>\n");
    return -1;
  }

  Mat image;
  image = imread( argv[1], 1 );

  if ( !image.data )
  {
    printf("No image data \n");
    return -1;
  }

  int *h_a, *d_a;
  float *h_b, *d_b;
  int img_size = image.rows * image.cols;

  h_a = (int *)malloc(img_size * sizeof(int));
  h_b = (float *)malloc(img_size * sizeof(float));

  hipMalloc((void **) &d_a, img_size * sizeof(int));
  hipMalloc((void **) &d_b, img_size * sizeof(float));

  hipMemcpy(d_a, h_a, img_size * sizeof(int), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(32, 32);
  dim3 numBlocks((int)ceil((float)image.cols/threadsPerBlock.x), (int)ceil((float)image.rows/threadsPerBlock.y));

  gpuGrayScale<<<numBlocks, threadsPerBlock>>>(d_a, d_b, image.cols, image.rows);
  cout << "im here" << endl;
  hipMemcpy(h_b, d_b, img_size, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);

  //namedWindow("Display Image", WINDOW_AUTOSIZE );
  //imshow("Display Image", image);
  
  //Mat img = (Mat_<float>(image.rows, image.cols) << h_b);
  //img = h_b;
  /*
  float r,g,b;
  for(int y=0;y<image.rows;y++){
    for(int x=0;x<image.cols;x++){
      // get pixel
      Vec3b color = img.at<Vec3b>(Point(x,y));

      r = color[0];
      g = color[1];
      b = color[2];

      //I = .299f * R + .587f * G + .114f * B
      color[2] = (r * 0.299 + g * 0.587 + b * 0.114);
      color[1] = (r * 0.299 + g * 0.587 + b * 0.114);
      color[0] = (r * 0.299 + g * 0.587 + b * 0.114);

      // set pixel
      img.at<Vec3b>(Point(x,y)) = color;
    }
  }
  */
  //imwrite("lena_out.jpg", img);

  //cout << h_b[0] << endl;

  //waitKey(0);

  return 0;
}