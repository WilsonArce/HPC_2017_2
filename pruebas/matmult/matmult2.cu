
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define MWIDTH 4096
#define MTILE 16
#define BWIDTH 16

__global__ void gpu_matrixMul(int *a, int *b, int *c, int Width, int tile_width){

  int start_row = (blockDim.y*blockIdx.y + threadIdx.y)*tile_width;
  int end_row = start_row + tile_width;
  int start_col = (blockDim.x*blockIdx.x + threadIdx.x)*tile_width;
  int end_col = start_col + tile_width;

  for (int row = start_row; row < end_row; row++) {
    for(int col = start_col; col < end_col; col++) {
      float sum = 0;
      for (int k = 0; k < Width; k++) {
        sum += a[row * Width + k]*b[k * Width + col];
      }
      c[row*Width+col] = sum;
    }
  }
}



int main(){
  double timeGPU;
  int *h_a, *h_b, *h_c, *d_a, *d_b, *d_c;
  h_a = (int *)malloc(MWIDTH*MWIDTH*sizeof(int));
  h_b = (int *)malloc(MWIDTH*MWIDTH*sizeof(int));
  h_c = (int *)malloc(MWIDTH*MWIDTH*sizeof(int));
  hipMalloc(&d_a, MWIDTH*MWIDTH*sizeof(int));
  hipMalloc(&d_b, MWIDTH*MWIDTH*sizeof(int));
  hipMalloc(&d_c, MWIDTH*MWIDTH*sizeof(int));

  for (int i = 0; i < MWIDTH*MWIDTH; i++) {
    h_a[i] = 1;
    h_b[i] = 1;
    h_c[i] = 0;}

  hipMemcpy(d_a, h_a, MWIDTH*MWIDTH*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, MWIDTH*MWIDTH*sizeof(int), hipMemcpyHostToDevice);
  hipMemset(d_c, 0, MWIDTH*MWIDTH*sizeof(int));

  clock_t startGPU  = clock();
  gpu_matrixMul<<<dim3((MWIDTH/(MTILE*BWIDTH)), (MWIDTH/(MTILE*BWIDTH))), dim3(BWIDTH,BWIDTH)>>>(d_a, d_b, d_c, MWIDTH, MTILE);

  hipMemcpy(h_c, d_c, MWIDTH*MWIDTH*sizeof(int), hipMemcpyDeviceToHost);
  timeGPU = ((double)(clock() - startGPU))/CLOCKS_PER_SEC;

  printf("tiempo GPU = %f s\n",timeGPU);

  /*
  for (int i=0; i < MWIDTH*MWIDTH; i++)
    if (h_c[i] != MWIDTH) {printf("Mismatch at offset %d, was: %d, should be: %d\n", i, h_c[i], MWIDTH); return 1;}
  printf("Success!\n");
  */

  return 0;
}