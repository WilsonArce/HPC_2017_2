#include <stdio.h>
#include <time.h>

using namespace cv;
using namespace std;

int main(int argc, char** argv )
{
	if ( argc != 2 )
    {
        printf("usage: DisplayImage.out <Image_Path>\n");
        return -1;
    }

    Mat image, image_gray;
		Mat grad;
		//char* window_name = "Sobel Demo - Simple Edge Detector";
		int scale = 1;
		int delta = 0;
		int ddepth = CV_16S;

    image = imread( argv[1], 1 );

    if ( !image.data )
    {
        printf("No image data \n");
        return -1;
    }
    //namedWindow("Display Image", WINDOW_AUTOSIZE );

    cvtColor( image, image_gray, CV_BGR2GRAY );

    /// Generate grad_x and grad_y
		Mat grad_x, grad_y;
		Mat abs_grad_x, abs_grad_y;

		/// Gradient X
		//Scharr( src_gray, grad_x, ddepth, 1, 0, scale, delta, BORDER_DEFAULT );
		Sobel( image_gray, grad_x, ddepth, 1, 0, 3, scale, delta, BORDER_DEFAULT );
		convertScaleAbs( grad_x, abs_grad_x );

		/// Gradient Y
		//Scharr( src_gray, grad_y, ddepth, 0, 1, scale, delta, BORDER_DEFAULT );
		Sobel( image_gray, grad_y, ddepth, 0, 1, 3, scale, delta, BORDER_DEFAULT );
		convertScaleAbs( grad_y, abs_grad_y );

		/// Total Gradient (approximate)
		addWeighted( abs_grad_x, 0.5, abs_grad_y, 0.5, 0, grad );

    imwrite("lena_out.jpg", grad);

    //imshow("Display Image", grad);

    cout << grad.size() << endl;

    //waitKey(0);

    return 0;
}
